
#include <hip/hip_runtime.h>
struct ArrayListNewCudaPixelData
{
	__device__  ArrayListNewCudaPixelData()
	{
	}
	unsigned char blue;
	unsigned char green;
	unsigned char red;
	unsigned char alpha;
};


// H264Images.ArrayListNewCuda
extern "C" __global__  void calGPU( unsigned char* dev_bitmap1, int dev_bitmap1Len0,  unsigned char* dev_bitmap2, int dev_bitmap2Len0,  unsigned char* dev_result, int dev_resultLen0,  int* imageWidth, int imageWidthLen0,  int* count, int countLen0,  int* possition, int possitionLen0);

// H264Images.ArrayListNewCuda
extern "C" __global__  void calGPU( unsigned char* dev_bitmap1, int dev_bitmap1Len0,  unsigned char* dev_bitmap2, int dev_bitmap2Len0,  unsigned char* dev_result, int dev_resultLen0,  int* imageWidth, int imageWidthLen0,  int* count, int countLen0,  int* possition, int possitionLen0)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	__shared__ int array[2];

	int arrayLen0 = 2;
	array[(0)] = 0;
	array[(1)] = 0;
	for (i = 0; i < imageWidth[(1)]; i++)
	{
		for (j = 0; j < imageWidth[(0)]; j++)
		{
			int num = (i * imageWidth[(0)] + j) * 4;
			ArrayListNewCudaPixelData pixelData = ArrayListNewCudaPixelData();
			ArrayListNewCudaPixelData pixelData2 = ArrayListNewCudaPixelData();
			pixelData.red = dev_bitmap1[(num + 2)];
			pixelData.green = dev_bitmap1[(num + 1)];
			pixelData.blue = dev_bitmap1[(num)];
			pixelData.alpha = dev_bitmap1[(num + 3)];
			pixelData2.green = dev_bitmap2[(num + 1)];
			pixelData2.red = dev_bitmap2[(num + 2)];
			pixelData2.blue = dev_bitmap2[(num)];
			pixelData2.alpha = dev_bitmap2[(num + 3)];
			bool flag = pixelData.red > pixelData2.red;
			int num2;
			if (flag)
			{
				num2 = (int)(pixelData.red - pixelData2.red);
			}
			else
			{
				num2 = (int)(pixelData2.red - pixelData.red);
			}
			bool flag2 = pixelData.alpha > pixelData2.alpha;
			int num3;
			if (flag2)
			{
				num3 = (int)(pixelData.alpha - pixelData2.alpha);
			}
			else
			{
				num3 = (int)(pixelData2.alpha - pixelData.alpha);
			}
			bool flag3 = pixelData.green > pixelData2.green;
			int num4;
			if (flag3)
			{
				num4 = (int)(pixelData.green - pixelData2.green);
			}
			else
			{
				num4 = (int)(pixelData2.green - pixelData.green);
			}
			bool flag4 = pixelData.blue > pixelData2.blue;
			int num5;
			if (flag4)
			{
				num5 = (int)(pixelData.blue - pixelData2.blue);
			}
			else
			{
				num5 = (int)(pixelData2.blue - pixelData.blue);
			}
			bool flag5 = num2 > 8 || num3 > 8 || num4 > 8 || num5 > 8;
			if (flag5)
			{
				 int* expr_1F9_cp_0 = array;
				int expr_1F9_cp_1 = 1;
				int num6 = expr_1F9_cp_0[(expr_1F9_cp_1)];
				expr_1F9_cp_0[(expr_1F9_cp_1)] = num6 + 1;
				possition[(num6)] = i;
				 int* expr_20F_cp_0 = array;
				int expr_20F_cp_1 = 1;
				num6 = expr_20F_cp_0[(expr_20F_cp_1)];
				expr_20F_cp_0[(expr_20F_cp_1)] = num6 + 1;
				possition[(num6)] = j;
				 int* expr_224_cp_0 = array;
				int expr_224_cp_1 = 0;
				num6 = expr_224_cp_0[(expr_224_cp_1)];
				expr_224_cp_0[(expr_224_cp_1)] = num6 + 1;
				dev_result[(num6)] = pixelData2.blue;
				 int* expr_23F_cp_0 = array;
				int expr_23F_cp_1 = 0;
				num6 = expr_23F_cp_0[(expr_23F_cp_1)];
				expr_23F_cp_0[(expr_23F_cp_1)] = num6 + 1;
				dev_result[(num6)] = pixelData2.green;
				 int* expr_25A_cp_0 = array;
				int expr_25A_cp_1 = 0;
				num6 = expr_25A_cp_0[(expr_25A_cp_1)];
				expr_25A_cp_0[(expr_25A_cp_1)] = num6 + 1;
				dev_result[(num6)] = pixelData2.red;
				 int* expr_275_cp_0 = array;
				int expr_275_cp_1 = 0;
				num6 = expr_275_cp_0[(expr_275_cp_1)];
				expr_275_cp_0[(expr_275_cp_1)] = num6 + 1;
				dev_result[(num6)] = pixelData2.alpha;
				count[(1)] = array[(1)];
				count[(0)] = array[(0)];
			}
		}
	}
}
